#include "hip/hip_runtime.h"


#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>


//tree node
struct Node {

	char data;
	unsigned freq;
	struct Node *left; *right;
}:

//holds Nodes
struct Tree {
	unsigned size;
	unsigned capacity
	struct Node**array;
};


//make new node
struct Tree* newNode(char data, unsigned freq) 
{ 
	struct Tree* temp 
		= (struct Tree*)malloc
(sizeof(struct Tree)); 

	temp->left = temp->right = NULL; 
	temp->data = data; 
	temp->freq = freq; 

	return temp; 
}



struct *Tree buildLeaves(char data[], int freq[], int size){
	struct *Tree tree = {size, size};
	for (int i = 0; i < size; ++i)
		tree->array[i] = newNode(data[i], freq[i]);

	tree->size = size;

	return tree;

}

__global__ void findMin(struct *Tree t, struct Node* min){
	Node *ptr1, *ptr2;
	ptr2 = t.array
	min = ptr2.freq
	for (ptr1 = t.array; *ptr1 != 0; ptr1++){
		if (ptr1.freq < min)
			min = ptr1
	}
}

void Insert(struct *Tree, struct Node *p){
	//
}

struct Node* buildTree(char letters[], int freq[], int size){
	struct Node *left, *right, *parent;	//set to null
	
	//build tree starting bottom up
	struct Tree* tree = buildLeaves(data, freq, size);
	struct Tree* t;	

	while (Tree.size != 1){

		//gpu functions
		hipMalloc(&t, size)
		hipMemcpy(t, tree,size, cudaMemcpyHosttoDevice);

		hipMalloc(&l, size)
		hipMemcpy(l, left, size, cudaMemcpyHosttoDevice);

		hipMalloc(&r, size)
		hipMemcpy(r, right, size, cudaMemcpyHosttoDevice);		

		findMin<<<2, 20>>>(t, struct Node* l);
		findMin<<<2, 20>>>(t, struct Node* r);

		hipMemcpy(left, l, size, cudaMemcpyDevicetoHost);
		hipMemcpy(right, r, size, cudaMemcpyDevicetoHost);
		
		parent = newNode('#', left->freq + right->freq);
		
		parent->left = left;
		parent->right = right;

		Insert(tree, parent);

	}
	
	findMin(tree, struct Node* root);

}

int Leaf(struct Tree* root){
	return !(root->left) && !(root->right);
}

void printCode(int code[], int a){
	for( int i = 0; i < a; ++i) printf("%d", code[i]);
	printf("\n")
}

void printAllCodes(struct Tree* root, int code[], int parent){
	if (root->left){
		code[parent] = 0;
		printAllCodes(root->left, code, parent + 1);
	}

	if(root->right){
		code[parent] = 1;
		printAllCodes(root->right, code, parent + 1);
	}
	if(Leaf(root)){
		printf("%c ", root->letter);
		printCode(code, parent);
	}

}

void Huffman(char letters[], int freq[], int size){
	//build tree
	struct Tree* root = buildTree(letters, freq, size);

	//print code
	int code[MAX_TREE_HT], parent = 0;

	printAllCodes(root, code, parent);
}


int main(){
	
	char letters[] = {a, b, c, d, e, f, g}
	int fq[] = {10, 42, 22, 30, 4, 58, 67}
	int size = sizeof(letters) / sizeof(letters[0]);
	Huffman(letters, fq, size)
	return 0;

}

	



