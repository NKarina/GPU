#include <stdio.h>
#include <hip/hip_runtime.h>

#include <hiprand/hiprand_kernel.h>

__global__ void multiply(float* Md, float* Nd, float* Pd, int Width){

        //int Row = blockIdx.y * blockDim.y + threadIdx.y;
        //int Col = blockIdx.x * blockDim.x + threadIdx.x;

        float Pvalue = 0;
        for (int k = 0; k < Width; ++k){
                Pvalue += Md[threadIdx.y*Width+k] * Nd[k*Width+threadIdx.x];
        }
        Pd[threadIdx.y*Width+threadIdx.x] = Pvalue;
}

int main(){
        float t= 0;
        hipEvent_t start, stop;

        hipEventCreate(&start);
        hipEventCreate(&stop);

        //HANDLE_ERROR(cudaEventRecord(start, 0));

        //determines properties of matrix i.e. nxn matrix
        int n = 16;

        float A[(n * n)], B[(n *n)], C[(n * n)];
        
        int size = (n * n) * sizeof(float);
        float *d_a, *d_b, *d_c;

        hipMalloc((void **) &d_a, size);
        hipMalloc((void **) &d_b, size);
        hipMalloc((void **) &d_c, size);

        for(int i =0; i< (n * n); i++){
                if (i%2 == 0){
                        A[i] = 0;
                        B[i] = 1;
                }
        }

        hipMemcpy(d_a, A, size, hipMemcpyHostToDevice);
        hipMemcpy(d_b, B, size, hipMemcpyHostToDevice);

        hipEventRecord(start, 0);

        multiply<<<1,256>>> (d_a, d_b, d_c, n);

        hipMemcpy(C, d_c, size, hipMemcpyDeviceToHost);

        hipFree(d_c);
        hipFree(d_b);
        hipFree(d_a);

        hipEventRecord(stop, 0);
        hipEventSynchronize (stop);
        hipEventElapsedTime(&t, start, stop);
       
        printf("Time:  %.2f ms \n", t);

        return 0;
}
