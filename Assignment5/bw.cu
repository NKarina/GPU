#include "hip/hip_runtime.h"
//Assignment 5


#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#define TC 256

__device__ bool comp(char *l, char *r, int width){
        char * tmp;
        for(int i = 0; i < width; i++){
                if (l[i] > r[i]){
                        tmp = r;
                        r = l;
                        l = tmp;
                        return true;
                }
        }
        return false;
}

__global__ void rotate(char *N, char **P, int width){
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        //float Pvalue = 0;
        char *current = N;
        P[0] = current;
        char *tmp;
        char ll;
        for(int j = 1; j < width; j++){
                ll = current[width - 1];
                tmp[i] = ll;
                for (int w = 0; w < width; w++){
                        tmp[w + 1] = current[w];
                }
                P[j + i] = tmp;
                current = tmp;
        }
}

__global__ void sort(char **N, char *P, int width){
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        //char ** current = N;
        bool c1, c2;
        for(int j = 0; j < width; j + 2){
                c1 = comp(N[j + i], N[j+ 1 + i], width);
        }
        for (int j = 1; j < width; j + 2){
                c2 = comp(N[j + i], N[j+1 + i], width);
        }
        while (c1 || c2){
                for (int j = 0; j < width; j + 2){
                        c1 = comp(N[i + j], N[i + j+1], width);
                }
                for (int j = 1; j < width; j + 2){
                        c2 = comp(N[i + j], N[i + j+1], width);
                }
        }
        //take last letters
        for (int j = 0; j < width; j++){
                P[i] = N[i + j][width - 1];
        }
}


int main(){
        float t = 0;
        hipEvent_t start, stop;


        hipEventCreate(&start);
        hipEventCreate(&stop);

       int width = 6;
        char I[width] = "orange";
        char O[width*width], FO[width];

        int size = width * sizeof(char);
        int bsize = width * width * sizeof(char);

        char *a, **o, *fo;

        hipMalloc((void **) &a, size);
        hipMalloc((void **) &o, bsize);
        hipMalloc((void **) &fo, size);

        hipMemcpy(a, I, size, hipMemcpyHostToDevice);

        hipEventRecord(start, 0);

        rotate<<<6,TC>>>(a, o, 6);

        hipMemcpy(O, o, bsize, hipMemcpyDeviceToHost);
        sort<<<6,TC>>>(o, fo, 6);

        hipMemcpy(FO, fo, size, hipMemcpyDeviceToHost);

        hipFree(a);
        hipFree(o);
        hipFree(fo);

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&t, start, stop);

        printf("Time: %.2f ms \n", t);
        return 0;

}

