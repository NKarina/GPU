//Assignment 4- Naive


#include <stdio.h>
#include <hip/hip_runtime.h>

#include <hiprand/hiprand_kernel.h>


__global__ void convolution_1D(float *N, float *M, float *P, int Mask_width, int Width){
        int i = blockIdx.x*blockDim.x + threadIdx.x;

        float Pvalue = 0;
        int N_start_point = i - (Mask_width/2);
        for (int j = 0; j< Mask_width; j++){
                if(N_start_point + j >= 0 && N_start_point + j < Width){
                        Pvalue += N[N_start_point + j]*M[j];
                }
        }
        P[i] = Pvalue;
}

int main(){
        float t = 0;
        hipEvent_t start, stop;

        hipEventCreate(&start);
        hipEventCreate(&stop);

        int n = 16;
        float In[n], O[n], M[(n/2 - 1)];

        int size = n  * sizeof(float);
        int msize = (n/2 - 1) * sizeof(float);
        float *d_i, *d_m, *d_o;

        hipMalloc((void **) &d_i, size);
        hipMalloc((void **) &d_m, msize);
        hipMalloc((void **) &d_o, size);

        for(int i = 0; i < n; i++){
                if (i < n/2 - 1){
                        M[i] = (i * 7) % 9;
                }
                In[i] = (i * 7) % 10;
        }

        hipMemcpy(d_i, In, size, hipMemcpyHostToDevice);
        hipMemcpy(d_m, M, msize, hipMemcpyHostToDevice);

        hipEventRecord(start, 0);

        convolution_1D<<<1, 2>>> (d_i, d_m, d_o, n/2 - 1, n);

        hipMemcpy(O, d_o, size, hipMemcpyDeviceToHost);

        hipFree(d_o);
        hipFree(d_m);
        hipFree(d_i);


        hipEventRecord(stop, 0);
        hipEventSynchronize (stop);
        hipEventElapsedTime(&t, start, stop);

        printf("Time: %.2f ms \n", t);

        return 0;

}
