#include "hip/hip_runtime.h"
/Assignment 4- constant


#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#define MAX_MASK_WIDTH 7
__constant__ float M[MAX_MASK_WIDTH];

__global__ void convo(float *N, float *P, int Mask_Width, int Width){
        int i = blockIdx.x*blockDim.x + threadIdx.x;
        float Pvalue = 0;
        int N_start_point = i - (Mask_Width/2);
        for (int j = 0; j < Mask_Width; j++){
                if(N_start_point + j >= 0 && N_start_point + j < Width){
                        Pvalue += N[N_start_point + j]*M[j];
                }
        }
        P[i] = Pvalue;

}

//#define MAX_MASK_WIDTH 7
//__constant__ float M[MAX_MASK_WIDTH];

int main(){
        float t = 0;
        hipEvent_t start, stop;

        hipEventCreate(&start);
        hipEventCreate(&stop);

        int n = 16;
        int msize = MAX_MASK_WIDTH * sizeof(float);
        float In[n], O[n];

        int size = n * sizeof(float);

        float *d_i, *d_o, *d_m;

        hipMalloc((void **) &d_i, size);
        hipMalloc((void **) &d_o, size);
        hipMalloc((void **) &d_m, msize);

        for(int i = 0; i < n; i++){
                if (i < n/2 -1){
                        M[i] = (i * 7) % 9;
                }
                In[i] = (i * 7) % 10;
        }

        hipMemcpy(d_i, In, size, hipMemcpyHostToDevice);
        hipMemcpyToSymbol(HIP_SYMBOL(d_m), M, msize);
        
        
        hipEventRecord(start, 0);

        convo<<<1, 2>>> (d_i, d_o, n/2 - 1, n);

        hipMemcpy(O, d_o, size, hipMemcpyDeviceToHost);


        hipFree(d_o);
        hipFree(d_m);
        hipFree(d_i);

        hipEventRecord(stop, 0);
        hipEventSynchronize (stop);
        hipEventElapsedTime(&t, start, stop);

        printf("Time: %.2f ms \n", t);

        return 0;

}


