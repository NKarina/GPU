//Assignment 4- shared


#include <stdio.h>
#include <hip/hip_runtime.h>

#include <hiprand/hiprand_kernel.h>

#define TILE_SIZE 7
#define MAX_MASK_WIDTH 7
__constant__ float M[MAX_MASK_WIDTH];

__global__ void convo(float *N, float *P, int Mask_Width, int Width){
        int i = blockIdx.x*blockDim.x + threadIdx.x;
        float Pvalue = 0;

        __shared__ float N_ds[TILE_SIZE];
        N_ds[threadIdx.x] = N[i];
        __syncthreads();

        int This_tile_start_point = blockIdx.x * blockDim.x;
        int Next_tile_start_point = (blockIdx.x + 1)*blockDim.x;
        int N_start_point = i - (Mask_Width/2);

        for (int j = 0; j < Mask_Width; j++){
                int N_index = N_start_point + j;

                if(N_index >= 0 && N_index < Width) {
                        if((N_index >= This_tile_start_point) && (N_index < Next_tile_start_point)) {
                                Pvalue += N_ds[threadIdx.x+j-(Mask_Width/2)]*M[j];
                        }
                        else{
                                Pvalue += N[N_index] * M[j];
                        }
                }
        }

}

int main(){
        float t = 0;
        hipEvent_t start, stop;

        hipEventCreate(&start);
        hipEventCreate(&stop);

        int n = 16;
        int msize = MAX_MASK_WIDTH * sizeof(float);
        float In[n], O[n];

        int size = n * sizeof(float);

        float *d_i, *d_o, *d_m;

        hipMalloc((void **) &d_i, size);
        hipMalloc((void **) &d_o, size);
        hipMalloc((void **) &d_m, msize);

        for(int i = 0; i < n; i++){
                if( i < n/2 - 1){
                        M[i] = (i * 7) % 9;
                }
                In[i] = (i * 7) % 10;
        }

        hipMemcpy(d_i, In, size, hipMemcpyHostToDevice);
        hipMemcpyToSymbol(HIP_SYMBOL(d_m), M, msize);

        hipEventRecord(start, 0);

        convo<<<1, 2>>> (d_i, d_o, n/2 - 1, n);

        hipMemcpy(O, d_o, size, hipMemcpyDeviceToHost);

        hipFree(d_o);
        hipFree(d_m);
        hipFree(d_i);
        hipEventRecord(stop, 0);
        hipEventSynchronize (stop);
        hipEventElapsedTime(&t, start, stop);

        printf("Time: %.2f ms \n", t);

        return 0;

}

